#include "hip/hip_runtime.h"
/*!
**************************************************************************************************
* InternImage
* Copyright (c) 2022 OpenGVLab
* Licensed under The MIT License [see LICENSE for details]
**************************************************************************************************
* Modified from
*https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/tree/pytorch_1.0.0
**************************************************************************************************
*/

#include "cuda/dcn1Dv3_im2col_cuda.cuh"
#include <vector>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/torch.h>

at::Tensor dcn1Dv3_cuda_forward(const at::Tensor &input, const at::Tensor &offset,
                              const at::Tensor &mask, const int kernel_l,
                              const int stride_l, const int pad_l,
                              const int dilation_l, const int group,
                              const int group_channels,
                              const float offset_scale, const int im2col_step) {
    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(offset.is_contiguous(), "offset tensor has to be contiguous");
    AT_ASSERTM(mask.is_contiguous(), "mask tensor has to be contiguous");
    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");

    const int batch = input.size(0);
    const int length_in = input.size(1);
    const int channels = input.size(2);
    const int length_out = (length_in + 2 * pad_l - (dilation_l * (kernel_l - 1) + 1)) / stride_l + 1;
    const int im2col_step_ = std::min(batch, im2col_step);

    AT_ASSERTM(batch % im2col_step_ == 0,
               "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);
    AT_ASSERTM(
        channels == (group * group_channels),
        "Input channels and group times group channels wont match: (%d vs %d).",
        channels, group * group_channels);

    auto output =
        at::zeros({batch, length_out, group * group_channels}, input.options());

    const int batch_n = im2col_step_;
    auto output_n = output.view({batch / batch_n, batch_n, length_out, group * group_channels});
    auto per_input_size = length_in * group * group_channels;
    auto per_offset_size = length_out * group * kernel_l * 2;
    auto per_mask_size = length_out * group * kernel_l;

    for (int n = 0; n < batch / im2col_step_; ++n) {
        auto columns = output_n.select(0, n);
        // AT_DISPATCH_FLOATING_TYPES(
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
            input.type(), "ms_deform_attn_forward_cuda", ([&] {
                dcn1Dv3_im2col_cuda(
                    at::cuda::getCurrentCUDAStream(),
                    input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                    offset.data<scalar_t>() +
                        n * im2col_step_ * per_offset_size,
                    mask.data<scalar_t>() + n * im2col_step_ * per_mask_size,
                    columns.data<scalar_t>(), kernel_l, stride_l, pad_l, dilation_l, group,
                    group_channels, batch_n, length_in, length_out, width_out, offset_scale);
            }));
    }

    return output;
}

std::vector<at::Tensor>
dcn1Dv3_cuda_backward(const at::Tensor &input, const at::Tensor &offset,
                      const at::Tensor &mask, const int kernel_l,
                      const int kernel_l, const int stride_l,
                      const int pad_l, const int dilation_l, const int group,
                      const int group_channels, const float offset_scale,
                      const at::Tensor &grad_output, const int im2col_step) {

    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(offset.is_contiguous(), "offset tensor has to be contiguous");
    AT_ASSERTM(mask.is_contiguous(), "mask tensor has to be contiguous");
    AT_ASSERTM(grad_output.is_contiguous(),
               "grad_output tensor has to be contiguous");
    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");
    AT_ASSERTM(grad_output.type().is_cuda(),
               "grad_output must be a CUDA tensor");

    const int batch = input.size(0);
    const int length_in = input.size(1);
    const int channels = input.size(2);
    const int length_out = (length_in + 2 * pad_l - (dilation_l * (kernel_l - 1) + 1)) / stride_l + 1;
    const int im2col_step_ = std::min(batch, im2col_step);

    AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);
    AT_ASSERTM(channels == (group * group_channels), "Input channels and group times group channels wont match: (%d vs %d).", channels, group * group_channels);

    auto dtype = input.dtype();
    if (dtype == at::kHalf) {
        dtype = at::kFloat;
    }

    auto grad_input = at::zeros_like(input, dtype);
    auto grad_offset = at::zeros_like(offset, dtype);
    auto grad_mask = at::zeros_like(mask, dtype);
    const int batch_n = im2col_step_;

    auto per_input_size = length_in  * group * group_channels;
    auto per_offset_size = length_out  * group * kernel_l * 2;
    auto per_mask_size = length_out  * group * kernel_l;
    auto grad_output_n = grad_output.view({batch / im2col_step_, batch_n, length_out, group, group_channels});

    for (int n = 0; n < batch / im2col_step_; ++n) {
        auto grad_output_g = grad_output_n.select(0, n);
        // AT_DISPATCH_FLOATING_TYPES(
        AT_DISPATCH_FLOATING_TYPES_AND_HALF( input.type(), "ms_deform_attn_backward_cuda", ([&] {
                dcn1Dv3_col2im_cuda(
                    at::cuda::getCurrentCUDAStream(),
                    grad_output_g.data<scalar_t>(),
                    input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                    offset.data<scalar_t>() +
                        n * im2col_step_ * per_offset_size,
                    mask.data<scalar_t>() + n * im2col_step_ * per_mask_size,
                    kernel_l, stride_l, pad_l, dilation_l, group, group_channels, batch_n,
                    length_in, length_out, offset_scale,
                    grad_input.data<opmath_t>() +
                        n * im2col_step_ * per_input_size,
                    grad_offset.data<opmath_t>() +
                        n * im2col_step_ * per_offset_size,
                    grad_mask.data<opmath_t>() +
                        n * im2col_step_ * per_mask_size);
            }));
    }

    if (input.dtype() == torch::kHalf) {
        return {grad_input.to(torch::kHalf), grad_offset.to(torch::kHalf),
                grad_mask.to(torch::kHalf)};
    } else {
        return {grad_input, grad_offset, grad_mask};
    }
}